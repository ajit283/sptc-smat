#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <stdio.h>

#include "common.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define WARP_SIZE 32

#define NUM_STAGES 2

__global__ void mmaOBTKernelSparse(half *bcsrValuesA, int *bcsrRowPtrA,
                                   int *bcsrColIdxA, half *B, half *C, size_t M,
                                   size_t N, size_t K, size_t nonzeroBlocks,
                                   int *blockInfo,
                                   int *relativeBlockIndexMapping) {
  // mmaCBTKernel
  const size_t K_tiles = div_ceil(K, MMA_K);

  const size_t warp_row = blockIdx.y * MMA_M;
  const size_t warp_col = blockIdx.x * MMA_N;

  size_t blockRow = blockIdx.y;
  size_t blockCol = blockIdx.x;

  size_t colRegions = (K + MMA_K - 1) / (MMA_K);

  if (warp_row >= M || warp_col >= N) {
    return;
  }

  __shared__ half A_smem[NUM_STAGES][MMA_M][MMA_K];
  __shared__ half B_smem[NUM_STAGES][MMA_N][MMA_K];
  __shared__ half C_smem[MMA_M][MMA_N];

  const size_t lane_id = threadIdx.x % WARP_SIZE;

  uint32_t RA[NUM_STAGES][4];
  uint32_t RB[NUM_STAGES][2];

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  // Load all pipeline stages.
  for (int stage = 0; stage < NUM_STAGES; ++stage) {
    pipe.producer_acquire();

    size_t ptr = bcsrRowPtrA[blockRow] + stage;
    if (ptr < bcsrRowPtrA[blockRow + 1]) {
      size_t i = bcsrColIdxA[ptr] / MMA_K;
      // skip empty block
      size_t blockIndex = blockRow * colRegions + i;

      size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

      size_t A_size = MMA_M * MMA_K * sizeof(half);
      size_t B_size = MMA_N * MMA_K * sizeof(half);

      cuda::memcpy_async(
          ((int4 *)(&A_smem[stage][lane_id / 2][0]) + lane_id % 2),
          (((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2)),
          sizeof(int4), pipe);

      // For matrix B
      if (lane_id < MMA_N * 2) { // Original condition preserved
        cuda::memcpy_async(
            ((int4 *)(&B_smem[stage][lane_id / 2][0]) + lane_id % 2),
            ((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) +
             lane_id % 2),
            sizeof(int4), pipe);
      }

      uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
          &A_smem[stage][lane_id % 16][(lane_id / 16) * 8]);
      LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                  A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[stage][lane_id % 8][((lane_id / 8) % 2) * 8]);
      LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);

      pipe.producer_commit();
    }
  }

  uint32_t RC[2] = {0, 0};
  int stage = 0;
#pragma unroll
  for (size_t ptr = bcsrRowPtrA[blockRow]; ptr < bcsrRowPtrA[blockRow + 1];
       ptr++) {

    cuda::pipeline_consumer_wait_prior<NUM_STAGES - 1>(pipe);

    __syncthreads();

    uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
        &A_smem[stage][lane_id % 16][(lane_id / 16) * 8]);
    LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                A_smem_lane_addr);

    uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
        &B_smem[stage][lane_id % 8][((lane_id / 8) % 2) * 8]);
    LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);

    HMMA16816(RC[0], RC[1], RA[stage][0], RA[stage][1], RA[stage][2],
              RA[stage][3], RB[stage][0], RB[stage][1], RC[0], RC[1]);

    __syncthreads();

    // Release the consumed stage.
    pipe.consumer_release();

    // Pre-load data for `num_stages` into the future.
    pipe.producer_acquire();

    size_t stage_ptr = ptr + NUM_STAGES;

    if (stage_ptr < bcsrRowPtrA[blockRow + 1]) {

      size_t i = bcsrColIdxA[stage_ptr] / MMA_K;
      // skip empty block
      size_t blockIndex = blockRow * colRegions + i;

      size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

      size_t A_size = MMA_M * MMA_K * sizeof(half);
      size_t B_size = MMA_N * MMA_K * sizeof(half);

      cuda::memcpy_async(
          ((int4 *)(&A_smem[stage][lane_id / 2][0]) + lane_id % 2),
          (((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2)),
          sizeof(int4), pipe);

      // For matrix B
      if (lane_id < MMA_N * 2) { // Original condition preserved
        cuda::memcpy_async(
            ((int4 *)(&B_smem[stage][lane_id / 2][0]) + lane_id % 2),
            ((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) +
             lane_id % 2),
            sizeof(int4), pipe);
      }

      uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
          &A_smem[stage][lane_id % 16][(lane_id / 16) * 8]);
      LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                  A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[stage][lane_id % 8][((lane_id / 8) % 2) * 8]);
      LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);
    }

    pipe.producer_commit();

    stage = (stage + 1) % NUM_STAGES;
  }

  *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
  *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

  __syncthreads();

  if (lane_id < MMA_M) {
    *((int4 *)(&C[(warp_row + lane_id) * N + warp_col])) =
        *((int4 *)(&C_smem[lane_id][0]));
  }
}

void mmaOBTKernel(half *bcsrValuesA, int *bcsrRowPtrA, int *bcsrColIdxA,
                  half *B, half *C, size_t M, size_t N, size_t K,
                  size_t nonzeroBlocks, int *blockInfo,
                  int *relativeBlockIndexMapping) {
  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

  mmaOBTKernelSparse<<<grid, block>>>(bcsrValuesA, bcsrRowPtrA, bcsrColIdxA, B,
                                      C, M, N, K, nonzeroBlocks, blockInfo,
                                      relativeBlockIndexMapping);
}
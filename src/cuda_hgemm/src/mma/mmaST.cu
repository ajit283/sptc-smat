#include "hip/hip_runtime.h"


// #include <__clang_cuda_builtin_vars.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cstdio>
#include <stdio.h>

#include "common.h"
#include "ptx.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define WARP_SIZE 32

// 1 - 2:4 sparse block
// 2 - dense block

__global__ void mmaSTKernelSparse(half *bcsrValuesA, half *B, half *C, size_t M,
                                  size_t N, size_t K, size_t nonzeroBlocks,
                                  int *blockInfo,
                                  int *relativeBlockIndexMapping) {

  // mmaSTKernel
  const size_t K_tiles = div_ceil(K, MMA_K);

  const size_t warp_row = blockIdx.y * MMA_M;
  const size_t warp_col = blockIdx.x * MMA_N;

  size_t blockRow = blockIdx.y;
  size_t blockCol = blockIdx.x;

  size_t colRegions = (K + MMA_K - 1) / (MMA_K);

  size_t blockIndex = blockRow * colRegions + blockCol;

  if (warp_row >= M || warp_col >= N) {
    return;
  }

  const size_t lane_id = threadIdx.x % WARP_SIZE;

  __shared__ half C_smem[MMA_M][MMA_N];

  uint32_t RC[2] = {0, 0};

#pragma unroll
  for (size_t i = 0; i < K_tiles; ++i) {
    // skip empty block
    int sparsityInfo = blockInfo[blockIndex];
    // if (sparsityInfo == 0) {
    //   printf("zero block");
    // } else if (sparsityInfo == 1) {
    //   printf("sparse block");
    // } else if (sparsityInfo == 2) {
    //   printf("dense block");
    // } else {
    //   printf("unknown block");
    // }
    size_t blockIndex = blockRow * colRegions + i;
    if (blockInfo[blockIndex] == 0) {
      continue;
    }
    size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

    // _shared__ half C_smem[MMA_M][MMA_N];
    // if (sparsityInfo == 2) {
    if (sparsityInfo == 2
        // || sparsityInfo == 1
    ) {

      __shared__ half A_smem[MMA_M][MMA_K];
      __shared__ half B_smem[MMA_N][MMA_K];
      // __shared__ half C_smem[MMA_M][MMA_N];

      *((int4 *)(&A_smem[lane_id / 2][0]) + lane_id % 2) =
          *((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2);

      // print matrix

      if (lane_id < MMA_N * 2) {
        *((int4 *)(&B_smem[lane_id / 2][0]) + lane_id % 2) =
            *((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) +
              lane_id % 2);
      }

      __syncthreads();

      uint32_t RA[4];
      uint32_t RB[2];

      uint32_t A_smem_lane_addr =
          __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 8]);
      LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
      LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

      HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0],
                RC[1]);
    }

    else if (sparsityInfo == 1) {

      //------------------------------------------------------------------------------
      __shared__ half A_smem_test[MMA_M][MMA_K];

      *((int4 *)(&A_smem_test[lane_id / 2][0]) + lane_id % 2) =
          *((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2);
      __syncthreads();
      if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
        printf("begin print matrix \n");
        for (int i = 0; i < MMA_M; i++) {
          for (int j = 0; j < MMA_K; j++) {
            printf("%f ", __half2float(A_smem_test[i][j]));
          }
          printf("\n");
        }

        printf("\n\n\n");
        printf("end print matrix \n");
      }
      __syncthreads();

      // -----------------------------------------------------------------------------

      __shared__ half A_smem[MMA_M][MMA_K / 2];
      __shared__ half B_smem[MMA_N][MMA_K];
      // __shared__ half C_smem[MMA_M][MMA_N];
      __shared__ char Meta_smem[MMA_M][MMA_K / 8];

      half *src = // length 8
          ((half *)((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                          (lane_id / 2) * MMA_K]) +
                    lane_id % 2));

      half src_sparse[4];

      char *cur_meta = (Meta_smem[lane_id / 2]) + (lane_id % 2);

      // for (int i = 0; i < 2; ++i) {
      //    pair = *(src + i * 4);
      //   half non_zero = (pair.x != (half)0.0f) ? pair.x : pair.y;
      //   src_sparse[i] = non_zero;

      //   // Set the metadata bits
      //   char position = (pair.x != (half)0.0f) ? (i * 2) : (i * 2 + 1);
      //   *cur_meta |= (position & 0x3) << (i * 2);
      // }
      for (int j = 0; j < 2; ++j) {
        int cur_src_sparse = 0;
        src_sparse[0 + (2 * j)] = 0;
        src_sparse[1 + (2 * j)] = 0;
        for (int i = 0; i < 4; ++i) {
          if (src[i + (4 * j)] != (half)0.0f) {
            src_sparse[cur_src_sparse + (2 * j)] = src[i + (4 * j)];
            *cur_meta |= i << (2 * cur_src_sparse + (4 * j));
            cur_src_sparse++;
          }
        }
      }

      *((int2 *)(&A_smem[lane_id / 2][0]) + lane_id % 2) =
          *((int2 *)src_sparse);

      __syncthreads();
      if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
        printf("begin print matrix sparse \n");
        for (int i = 0; i < MMA_M; i++) {
          for (int j = 0; j < MMA_K / 2; j++) {
            printf("%f ", __half2float(A_smem[i][j]));
          }
          printf("\n");
        }

        printf("\n\n\n");
        printf("end print matrix sparse \n");
      }
      __syncthreads();

      __syncthreads();

      char metadata[4];

      metadata[0] = *cur_meta;
      metadata[1] = *(cur_meta + 1);
      metadata[2] = *(cur_meta + (2 * 8));
      metadata[2] = *(cur_meta + (2 * 8) + 1);

      // print metadata[0] as bits
      if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
        printf("begin print metadata as bits \n");
        for (int i = 0; i < 4; i++) {
          for (int bit = 7; bit >= 0; bit--) {
            printf("%d", (metadata[i] >> bit) & 1);
          }
          printf(" "); // Space between each char
        }
        printf("\n");
        printf("end print metadata as bits \n");
      }

      uint32_t RA[2];
      uint32_t RB[2];

      uint32_t A_smem_lane_addr =
          __cvta_generic_to_shared(&A_smem[lane_id % 16][(lane_id / 16) * 4]);
      LDMATRIX_X2(RA[0], RA[1], A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[lane_id % 8][((lane_id / 8) % 2) * 8]);
      LDMATRIX_X2(RB[0], RB[1], B_smem_lane_addr);

      // equivalent for metadata

      HMMA16816_SPARSE(RC[0], RC[1], RA[0], RA[1], RB[0], RB[1], RC[0], RC[1],
                       *(uint32_t *)metadata, 0x0);
    }

    __syncthreads();
  }

  *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
  *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

  __syncthreads();

  if (lane_id < MMA_M) {
    *((int4 *)(&C[(warp_row + lane_id) * N + warp_col])) =
        *((int4 *)(&C_smem[lane_id][0]));
  }
}

void mmaSTKernel(half *bcsrValuesA, half *B, half *C, size_t M, size_t N,
                 size_t K, size_t nonzeroBlocks, int *blockInfo,
                 int *relativeBlockIndexMapping) {
  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

  mmaSTKernelSparse<<<grid, block>>>(bcsrValuesA, B, C, M, N, K, nonzeroBlocks,
                                     blockInfo, relativeBlockIndexMapping);
}

#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <stdio.h>

#include "common.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 32

#define WARP_SIZE 32

#define NUM_STAGES 2

__global__ void mmaOBTSKernelSparse_large(half *bcsrValuesA, int *bcsrRowPtrA,
                                          int *bcsrColIdxA, char *metadata,
                                          half *sparseMatrixA, half *B, half *C,
                                          size_t M, size_t N, size_t K,
                                          size_t nonzeroBlocks, int *blockInfo,
                                          int *relativeBlockIndexMapping) {

  const size_t K_tiles = div_ceil(K, MMA_K);

  const size_t warp_row = blockIdx.y * MMA_M;
  const size_t warp_col = blockIdx.x * MMA_N;

  size_t blockRow = blockIdx.y;
  size_t blockCol = blockIdx.x;

  size_t colRegions = (K + MMA_K - 1) / (MMA_K);

  if (warp_row >= M || warp_col >= N) {
    return;
  }

  __shared__ half A_smem[NUM_STAGES][MMA_M][MMA_K];
  __shared__ half B_smem[NUM_STAGES][MMA_N][MMA_K];
  __shared__ half C_smem[MMA_M][MMA_N];

  __shared__ half A_smem_sparse[NUM_STAGES][MMA_M][MMA_K / 2];
  __shared__ half B_smem_sparse[NUM_STAGES][MMA_N][MMA_K];
  __shared__ char Meta_smem_sparse[NUM_STAGES][MMA_M][MMA_K / 8];

  const size_t lane_id = threadIdx.x % WARP_SIZE;

  uint32_t RA[NUM_STAGES][4];
  uint32_t RB[NUM_STAGES][4];

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  auto loadStages = [&] __device__(size_t stage_ptr, int stage) {
    if (stage_ptr < bcsrRowPtrA[blockRow + 1]) {

      size_t i = bcsrColIdxA[stage_ptr] / MMA_K;

      if (i >= 512) {
        printf("problem");
      }

      // skip empty block
      size_t blockIndex = blockRow * colRegions + i;

      size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

      size_t A_size = MMA_M * MMA_K * sizeof(half);
      size_t B_size = MMA_N * MMA_K * sizeof(half);

      int sparsityInfo = blockInfo[blockIndex];

      if (sparsityInfo == 2) {

        cuda::memcpy_async(
            ((long4 *)(&A_smem[stage][lane_id / 2][0]) + lane_id % 2),
            (((long4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * MMA_K +
                                     (lane_id / 2) * MMA_K]) +
              lane_id % 2)),
            sizeof(long4), pipe);

        // For matrix B
        if (lane_id < MMA_N * 2) { // Original condition preserved
          cuda::memcpy_async(
              ((long4 *)(&B_smem[stage][lane_id / 2][0]) + lane_id % 2),
              ((long4 *)(&B[i * MMA_K + (warp_col + lane_id / 2) * K]) +
               lane_id % 2),
              sizeof(long4), pipe);
        }

      } else if (sparsityInfo == 1) {

        cuda::memcpy_async(
            ((int4 *)(&A_smem_sparse[stage][lane_id / 2][0]) + lane_id % 2),
            (((int4 *)(sparseMatrixA)) + relativeIndex * MMA_M * (MMA_K / 16) +
             lane_id),
            sizeof(int4), pipe);
        cuda::memcpy_async(
            ((half *)(Meta_smem_sparse[stage][lane_id / 2]) + (lane_id % 2)),
            ((half *)metadata +
             (relativeIndex * MMA_M * (MMA_K / 16) + lane_id)),
            sizeof(half), pipe);

        cuda::memcpy_async(
            ((int4 *)(&B_smem_sparse[stage][lane_id / 4][0]) + lane_id % 4),
            ((int4 *)(&B[i * MMA_K + (warp_col + lane_id / 4) * K]) +
             lane_id % 4),
            sizeof(int4), pipe);
      }
    }
  };

  // Load all pipeline stages.
  for (int stage = 0; stage < NUM_STAGES; ++stage) {
    pipe.producer_acquire();

    size_t ptr = bcsrRowPtrA[blockRow] + stage;
    loadStages(ptr, stage);

    pipe.producer_commit();
  }

  uint32_t RC[2] = {0, 0};
  int stage = 0;
#pragma unroll
  for (size_t ptr = bcsrRowPtrA[blockRow]; ptr < bcsrRowPtrA[blockRow + 1];
       ptr++) {

    cuda::pipeline_consumer_wait_prior<NUM_STAGES - 1>(pipe);
    size_t i = bcsrColIdxA[ptr] / MMA_K;
    // skip empty block
    size_t blockIndex = blockRow * colRegions + i;

    size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

    size_t A_size = MMA_M * MMA_K * sizeof(half);
    size_t B_size = MMA_N * MMA_K * sizeof(half);

    int sparsityInfo = blockInfo[blockIndex];

    __syncthreads();

    if (sparsityInfo == 2) {

      uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
          &A_smem[stage][lane_id % 16][(lane_id / 16) * 8]);
      LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                  A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[stage][lane_id % 8][((lane_id / 8) % 2) * 8]);
      LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);

      HMMA16816(RC[0], RC[1], RA[stage][0], RA[stage][1], RA[stage][2],
                RA[stage][3], RB[stage][0], RB[stage][1], RC[0], RC[1]);

      A_smem_lane_addr = __cvta_generic_to_shared(
          &A_smem[stage][lane_id % 16][(lane_id / 16) * 8 + 16]);
      LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                  A_smem_lane_addr);

      B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem[stage][lane_id % 8][((lane_id / 8) % 2) * 8 + 16]);
      LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);

      HMMA16816(RC[0], RC[1], RA[stage][0], RA[stage][1], RA[stage][2],
                RA[stage][3], RB[stage][0], RB[stage][1], RC[0], RC[1]);

    } else if (sparsityInfo == 1) {

      uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
          &A_smem_sparse[stage][lane_id % 16]
                        [(lane_id / 16) * (MMA_K / 2 / 2)]);
      LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                  A_smem_lane_addr);

      uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
          &B_smem_sparse[stage][lane_id % 8]
                        [((lane_id / 8) % 2) * (MMA_K / 2)]);
      LDMATRIX_X4(RB[stage][0], RB[stage][1], RB[stage][2], RB[stage][3],
                  B_smem_lane_addr);

      char metadata_local[4];

      metadata_local[0] =
          (char)((Meta_smem_sparse[stage][lane_id / 4][0 + 2 * (lane_id % 2)]));
      metadata_local[1] =
          (char)((Meta_smem_sparse[stage][lane_id / 4][1 + 2 * (lane_id % 2)]));
      metadata_local[2] = (char)((
          Meta_smem_sparse[stage][(lane_id / 4) + 8][0 + 2 * (lane_id % 2)]));
      metadata_local[3] = (char)((
          Meta_smem_sparse[stage][(lane_id / 4) + 8][1 + 2 * (lane_id % 2)]));

      uint32_t meta_value;
      memcpy(&meta_value, metadata_local, sizeof(uint32_t));

      // HMMA16816_SPARSE(RC[0], RC[1], RA[stage][0], RA[stage][1],
      // RB[stage][0],
      //                  RB[stage][1], RC[0], RC[1], meta_value, 0x0);

      HMMA16832_SPARSE(RC[0], RC[1], RA[stage][0], RA[stage][1], RA[stage][2],
                       RA[stage][3], RB[stage][0], RB[stage][1], RB[stage][2],
                       RB[stage][3], RC[0], RC[1], meta_value, 0x0);
    }

    __syncthreads();

    // Release the consumed stage.
    pipe.consumer_release();

    // Pre-load data for `num_stages` into the future.
    pipe.producer_acquire();

    size_t stage_ptr = ptr + NUM_STAGES;

    loadStages(stage_ptr, stage);

    pipe.producer_commit();

    stage = (stage + 1) % NUM_STAGES;
  }

  *((uint32_t *)(&C_smem[lane_id / 4][0]) + lane_id % 4) = RC[0];
  *((uint32_t *)(&C_smem[lane_id / 4 + 8][0]) + lane_id % 4) = RC[1];

  __syncthreads();

  if (lane_id < MMA_M) {
    *((int4 *)(&C[(warp_row + lane_id) * N + warp_col])) =
        *((int4 *)(&C_smem[lane_id][0]));
  }
}

void mmaOBTSKernel_large(half *bcsrValuesA, int *bcsrRowPtrA, int *bcsrColIdxA,
                         char *metadata, half *sparseMatrixA, half *B, half *C,
                         size_t M, size_t N, size_t K, size_t nonzeroBlocks,
                         int *blockInfo, int *relativeBlockIndexMapping) {
  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

  mmaOBTSKernelSparse_large<<<grid, block>>>(
      bcsrValuesA, bcsrRowPtrA, bcsrColIdxA, metadata, sparseMatrixA, B, C, M,
      N, K, nonzeroBlocks, blockInfo, relativeBlockIndexMapping);
}
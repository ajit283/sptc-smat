#include "hip/hip_runtime.h"
#pragma once

#include <cmath>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>
#include <stdio.h>

#include "common.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define BLOCKSIZE 2

#define WARP_SIZE 32

#define NUM_STAGES 2

__global__ void mmaOBTKernelSparse_tiled(half *bcsrValuesA, int *bcsrRowPtrA,
                                         int *bcsrColIdxA, half *B, half *C,
                                         size_t M, size_t N, size_t K,
                                         size_t nonzeroBlocks, int *blockInfo,
                                         int *relativeBlockIndexMapping) {
  // mmaCBTKernel
  const size_t K_tiles = div_ceil(K, MMA_K);

  const size_t warp_row = blockIdx.y * MMA_M * BLOCKSIZE;
  const size_t warp_col = blockIdx.x * MMA_N * BLOCKSIZE;

  size_t blockRow = blockIdx.y * BLOCKSIZE;
  size_t blockCol = blockIdx.x * BLOCKSIZE;

  // size_t colRegions = (K + MMA_K - 1) / (MMA_K);
  size_t colRegions = (K + MMA_K * BLOCKSIZE - 1) / (MMA_K * BLOCKSIZE);

  if (warp_row >= M || warp_col >= N) {
    return;
  }

  int this_ptr = bcsrRowPtrA[blockRow];

  int next_ptr = bcsrRowPtrA[blockRow + 1];

  __shared__ half A_smem[NUM_STAGES][BLOCKSIZE][BLOCKSIZE][MMA_M][MMA_K];
  __shared__ half B_smem[NUM_STAGES][BLOCKSIZE][BLOCKSIZE][MMA_N][MMA_K];
  __shared__ half C_smem[BLOCKSIZE][BLOCKSIZE][MMA_M][MMA_N];

  const size_t lane_id = threadIdx.x % WARP_SIZE;

  // tileIdX should be in [0, BLOCKSIZE_X)
  const size_t tileIdX = (threadIdx.x / WARP_SIZE) % BLOCKSIZE;
  // tileIdY should be in [0, BLOCKSIZE_Y)
  const size_t tileIdY = threadIdx.x / (WARP_SIZE * BLOCKSIZE);

  uint32_t RA[NUM_STAGES][4];
  uint32_t RB[NUM_STAGES][2];

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  // Load all pipeline stages.
  for (int stage = 0; stage < NUM_STAGES; ++stage) {
    pipe.producer_acquire();

    size_t ptr = this_ptr + stage;
    // printf("ptr %d\n", ptr);
    // printf("next_ptr %d\n", next_ptr);
    if (ptr < next_ptr) {
      size_t i = bcsrColIdxA[ptr] / MMA_K;
      // skip empty block
      size_t blockIndex = blockRow * colRegions + i;

      size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

      size_t A_size = MMA_M * MMA_K * sizeof(half);
      size_t B_size = MMA_N * MMA_K * sizeof(half);

      // printf("before 1st memcpy_async");

      cuda::memcpy_async(
          ((int4 *)(&A_smem[stage][tileIdY][tileIdX][lane_id / 2][0]) +
           lane_id % 2),
          (((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * BLOCKSIZE * MMA_K *
                                      BLOCKSIZE +
                                  (tileIdY * MMA_K * MMA_M * BLOCKSIZE +
                                   tileIdX * MMA_M * MMA_K) +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2)),
          sizeof(int4), pipe);
      // printf("after 1st memcpy_async");

      // For matrix B
      if (lane_id < MMA_N * 2) { // Original condition preserved
        cuda::memcpy_async(
            ((int4 *)(&B_smem[stage][tileIdY][tileIdX][lane_id / 2][0]) +
             lane_id % 2),
            ((int4 *)(&B[i * MMA_K * BLOCKSIZE + tileIdX * MMA_K +
                         (warp_col + tileIdY * WARP_SIZE + lane_id / 2) * K]) +
             lane_id % 2),
            sizeof(int4), pipe);
      }

      pipe.producer_commit();
    }
  }

  uint32_t RC[2] = {0, 0};
  int stage = 0;
#pragma unroll
  for (size_t ptr = this_ptr; ptr < next_ptr; ptr++) {

    cuda::pipeline_consumer_wait_prior<NUM_STAGES - 1>(pipe);

    __syncthreads();

    uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
        &A_smem[stage][tileIdY][tileIdX][lane_id % 16][(lane_id / 16) * 8]);
    LDMATRIX_X4(RA[stage][0], RA[stage][1], RA[stage][2], RA[stage][3],
                A_smem_lane_addr);

    uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
        &B_smem[stage][tileIdY][tileIdX][lane_id % 8][((lane_id / 8) % 2) * 8]);
    LDMATRIX_X2(RB[stage][0], RB[stage][1], B_smem_lane_addr);

    HMMA16816(RC[0], RC[1], RA[stage][0], RA[stage][1], RA[stage][2],
              RA[stage][3], RB[stage][0], RB[stage][1], RC[0], RC[1]);

    __syncthreads();

    // Release the consumed stage.
    pipe.consumer_release();

    // Pre-load data for `num_stages` into the future.
    pipe.producer_acquire();

    size_t stage_ptr = ptr + NUM_STAGES;

    if (stage_ptr < next_ptr) {

      size_t i = bcsrColIdxA[stage_ptr] / MMA_K;
      // skip empty block
      size_t blockIndex = blockRow * colRegions + i;

      size_t relativeIndex = relativeBlockIndexMapping[blockIndex];

      size_t A_size = MMA_M * MMA_K * sizeof(half);
      size_t B_size = MMA_N * MMA_K * sizeof(half);

      cuda::memcpy_async(
          ((int4 *)(&A_smem[stage][tileIdY][tileIdX][lane_id / 2][0]) +
           lane_id % 2),
          (((int4 *)(&bcsrValuesA[(relativeIndex)*MMA_M * BLOCKSIZE * MMA_K *
                                      BLOCKSIZE +
                                  (tileIdY * MMA_K * MMA_M * BLOCKSIZE +
                                   tileIdX * MMA_M * MMA_K) +
                                  (lane_id / 2) * MMA_K]) +
            lane_id % 2)),
          sizeof(int4), pipe);

      // For matrix B
      if (lane_id < MMA_N * 2) { // Original condition preserved
        // cuda::memcpy_async(
        //     ((int4 *)(&B_smem[stage][tileIdY][tileIdX][lane_id / 2][0]) +
        //      lane_id % 2),
        //     ((int4 *)(&B[i * MMA_K * BLOCKSIZE + tileIdX * MMA_K +
        //                  (warp_col + lane_id / 2) * K]) +
        //      lane_id % 2),
        //     sizeof(int4), pipe);
        cuda::memcpy_async(
            ((int4 *)(&B_smem[stage][tileIdY][tileIdX][lane_id / 2][0]) +
             lane_id % 2),
            ((int4 *)(&B[i * MMA_K * BLOCKSIZE + tileIdX * MMA_K +
                         (warp_col + tileIdY * WARP_SIZE + lane_id / 2) * K]) +
             lane_id % 2),
            sizeof(int4), pipe);
      }
    }

    pipe.producer_commit();

    stage = (stage + 1) % NUM_STAGES;
  }

  *((uint32_t *)(&C_smem[tileIdY][tileIdX][lane_id / 4][0]) + lane_id % 4) =
      RC[0];
  *((uint32_t *)(&C_smem[tileIdY][tileIdX][lane_id / 4 + 8][0]) + lane_id % 4) =
      RC[1];

  __syncthreads();

  if (lane_id < MMA_M) {
    *((int4 *)(&C[(warp_row + (tileIdY * MMA_M) + lane_id) * N + warp_col +
                  tileIdX * MMA_N])) =
        *((int4 *)(&C_smem[tileIdY][tileIdX][lane_id][0]));
  }
}

void mmaOBTKernel_tiled(half *bcsrValuesA, int *bcsrRowPtrA, int *bcsrColIdxA,
                        half *B, half *C, size_t M, size_t N, size_t K,
                        size_t nonzeroBlocks, int *blockInfo,
                        int *relativeBlockIndexMapping) {
  dim3 block(WARP_SIZE * BLOCKSIZE * BLOCKSIZE);
  dim3 grid(div_ceil(N, MMA_N * BLOCKSIZE), div_ceil(M, MMA_M * BLOCKSIZE));

  mmaOBTKernelSparse_tiled<<<grid, block>>>(
      bcsrValuesA, bcsrRowPtrA, bcsrColIdxA, B, C, M, N, K, nonzeroBlocks,
      blockInfo, relativeBlockIndexMapping);
}